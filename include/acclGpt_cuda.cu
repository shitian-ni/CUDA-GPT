#include "hip/hip_runtime.h"
//
//  Created by Shitian Ni on 1/18/18.
//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include<iostream>
#include<string.h>
#include "parameter.h"

using namespace std;

__device__ double d_H[ROW_H][COL_H], d_Ht[ROW][COL_Ht];
__device__ unsigned char d_image1[1024][1024];
__device__ unsigned char d_image2[1024][1024];
__device__ double d_g[G_NUM], d_g_can1[ROW][COL], d_g_nor1[ROW][COL];
__device__ int d_g_ang1[ROW][COL];
__device__ char d_sHoG1[ROW - 4][COL - 4];




int iDivUp(int hostPtr, int b){ return ((hostPtr % b) != 0) ? (hostPtr / b + 1) : (hostPtr / b); };

//https://stackoverflow.com/a/14038590
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void Ht_1() {
	int margin = 2;
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((y >= ROW - 2 * margin) || (x >= 3 * 64 * (COL - 2 * margin))) {
        return;
    }

    d_Ht[y][x] =  d_H[y][x + (COL - 2 * margin) * 3 * 64 * 5];
};
__global__ void Ht_2() {
	int margin = 2;
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((y >= ROW - 2 * margin) || (x >= 3 * 64 * (COL - 2 * margin))) {
        return;
    }

    d_Ht[y][x] =  d_H[y][x];   
};
__global__ void Ht_3(int count, double newVar) {
	int margin = 2;
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((y >= ROW - 2 * margin) || (x >= 3 * 64 * (COL - 2 * margin))) {
        return;
    }
    double var_p_1 = pow(2.0,count + 1 -5);
    double var = var_p_1 / 2.0;
    d_Ht[y][x] = d_H[y][x + (COL - 2 * margin) * 3 * 64 * count] +
                                   (d_H[y][x + (COL - 2 * margin) * 3 * 64 * (count + 1)] - d_H[y][x + (COL - 2 * margin) * 3 * 64 * count])
                                 / (var_p_1 - var)
                                 * (newVar - var);                    
};


//1000 times 1200~1300ms
//http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
template<typename T>
__device__ void customAdd(T* sdata,T* g_odata){
 	int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;
	// do reduction in shared mem
	if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads();
	if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
	if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
	if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();
	if (tid < 32){ sdata[tid] += sdata[tid + 32]; }__syncthreads();
	if (tid < 16){ sdata[tid] += sdata[tid + 16]; }__syncthreads();
	if (tid < 8){ sdata[tid] += sdata[tid + 8]; }__syncthreads();
	if (tid < 4){ sdata[tid] += sdata[tid + 4]; }__syncthreads();
	if (tid < 2){ sdata[tid] += sdata[tid + 2]; }__syncthreads();
	if (tid < 1){ sdata[tid] += sdata[tid + 1]; }__syncthreads();
	// write result for this block to global mem
	if (tid == 0) {atomicAdd(g_odata        , sdata[tid]);}

}
__global__ void weightedAVG() {

	// __shared__ double sdata[TPB_X_TPB];
	__shared__ double sdata[6][TPB_X_TPB];

    int x1 = blockIdx.x*blockDim.x + threadIdx.x;
    int y1 = blockIdx.y*blockDim.y + threadIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;

    int margin = 2;

    double sHoGnumber[64] = sHoGNUMBER;
	double dx1=x1 - CX;
	double dy1=y1 - CY;
	bool condition = ((y1 >= margin) && (x1 >= margin) && (y1 < ROW-margin) && (x1 < COL-margin) && d_sHoG1[y1 - margin][x1 - margin] != -1);
	double t0 = 0;
	double tx2 = 0;
	double ty2 = 0;
	int thre = -1;
    for (int s = 0 ; condition && s < 64 ; s++) {
        if (d_sHoG1[y1 - margin][x1 - margin] == sHoGnumber[s]) {
            thre = s * 3 * (COL - 2 * margin);
            t0     = d_Ht[y1 - margin][thre + x1 - margin]                          * d_g_can1[y1][x1];
		    tx2    = d_Ht[y1 - margin][thre + x1 - margin + (COL - 2 * margin)]     * d_g_can1[y1][x1];
		    ty2    = d_Ht[y1 - margin][thre + x1 - margin + (COL - 2 * margin) * 2] * d_g_can1[y1][x1];
            break;
        }
    }
	

    sdata[0][tid]=t0; 
	sdata[1][tid]=tx2; 
	sdata[2][tid]=ty2; 
	sdata[3][tid]=t0  * dx1;
	sdata[4][tid]=t0  * dx1 * dx1; 
	sdata[5][tid]=t0  * dx1 * dx1 * dx1; 
	__syncthreads(); 

	customAdd(sdata[0],d_g); 
	customAdd(sdata[1],d_g+21);
	customAdd(sdata[2],d_g+22);
	customAdd(sdata[3],d_g+3);
	customAdd(sdata[4],d_g+4);
	customAdd(sdata[5],d_g+5);
	__syncthreads(); 

	sdata[0][tid]=t0  * dx1 * dx1 * dx1 * dx1; 
	sdata[1][tid]=t0  * dy1; 
	sdata[2][tid]=t0  * dy1 * dy1; 
	sdata[3][tid]=t0  * dy1 * dy1 * dy1;  
	sdata[4][tid]=t0  * dy1 * dy1 * dy1 * dy1; 
	sdata[5][tid]=t0  * dx1 * dy1; 
	__syncthreads(); 

	customAdd(sdata[0],d_g+6);
	customAdd(sdata[1],d_g+7);
	customAdd(sdata[2],d_g+8);
	customAdd(sdata[3],d_g+9);
	customAdd(sdata[4],d_g+10); 
	customAdd(sdata[5],d_g+11);
	__syncthreads(); 

	sdata[0][tid]=t0  * dx1 * dx1 * dy1; 
	sdata[1][tid]=t0  * dx1 * dx1 * dx1 * dy1; 
	sdata[2][tid]=t0  * dx1 * dy1 * dy1; 
	sdata[3][tid]=t0  * dx1 * dx1 * dy1 * dy1;
	sdata[4][tid]=t0  * dx1 * dy1 * dy1 * dy1;
	sdata[5][tid]=tx2 * dx1; 
	__syncthreads();  

	customAdd(sdata[0],d_g+12);
	customAdd(sdata[1],d_g+13);
	customAdd(sdata[2],d_g+14);
	customAdd(sdata[3],d_g+15);
	customAdd(sdata[4],d_g+16);
	customAdd(sdata[5],d_g+17);
	__syncthreads();  

	sdata[0][tid]=tx2 * dy1; 
	sdata[1][tid]=ty2 * dx1; 
	sdata[2][tid]=ty2 * dy1;   
	sdata[3][tid]=tx2 * dx1 * dx1; 
	sdata[4][tid]=ty2 * dx1 * dy1;  
	sdata[5][tid]=tx2 * dx1 * dy1; 
	__syncthreads();

	customAdd(sdata[0],d_g+18);
	customAdd(sdata[1],d_g+19);
	customAdd(sdata[2],d_g+20);
	customAdd(sdata[3],d_g+23);
	customAdd(sdata[4],d_g+24);
	customAdd(sdata[5],d_g+25);
	__syncthreads();

	sdata[0][tid]=ty2 * dy1 * dy1; 
	__syncthreads(); 
	customAdd(sdata[0],d_g+26); 
	__syncthreads();  
};

__global__ void cuda_roberts8() {
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((y >= ROW) || (x >= COL)) {
        return;
    }

	/* extraction of gradient information by Roberts operator */
	/* with 8-directional codes and strength */
	double delta_RD, delta_LD;
	double angle;

	/* angle & norm of gradient vector calculated
     by Roberts operator */

	if(y >= ROW-1 || x >= COL-1){
		d_g_ang1[y][x] = -1;
		d_g_nor1[y][x] = 0.0;
		return;
	}

	delta_RD = d_image1[y][x + 1] - d_image1[y + 1][x];
	delta_LD = d_image1[y][x]     - d_image1[y + 1][x + 1];
	d_g_nor1[y][x] = sqrt(delta_RD * delta_RD + delta_LD * delta_LD);

	if (d_g_nor1[y][x] == 0.0 || delta_RD * delta_RD + delta_LD * delta_LD < NoDIRECTION * NoDIRECTION) {
		d_g_ang1[y][x] = -1;
		return;
	}
	if (abs(delta_RD) == 0.0) {
		if (delta_LD > 0) d_g_ang1[y][x] = 3;
		else if (delta_LD < 0) d_g_ang1[y][x] = 7;
		else d_g_ang1[y][x] = -1;
		return;
	} 
	angle = atan2(delta_LD, delta_RD);
	if (     angle >  7.0 / 8.0 * PI) d_g_ang1[y][x] = 5;
	else if (angle >  5.0 / 8.0 * PI) d_g_ang1[y][x] = 4;
	else if (angle >  3.0 / 8.0 * PI) d_g_ang1[y][x] = 3;
	else if (angle >  1.0 / 8.0 * PI) d_g_ang1[y][x] = 2;
	else if (angle > -1.0 / 8.0 * PI) d_g_ang1[y][x] = 1;
	else if (angle > -3.0 / 8.0 * PI) d_g_ang1[y][x] = 0;
	else if (angle > -5.0 / 8.0 * PI) d_g_ang1[y][x] = 7;
	else if (angle > -7.0 / 8.0 * PI) d_g_ang1[y][x] = 6;
	else d_g_ang1[y][x] = 5;	
}

/*
	d_cuda_defcan_vars[0]:  mean
	d_cuda_defcan_vars[1]:  norm
	d_cuda_defcan_vars[2]:  npo
*/
__device__ double d_cuda_defcan_vars[3];
__global__ void cuda_defcan1() {
	int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((y >= ROW) || (x >= COL)) {
        return;
    }

	/* definite canonicalization */
	double ratio; // mean: mean value, norm: normal factor, ratio:
	int margine = CANMARGIN / 2;
	int condition = ((x>=margine && y>=margine) && 
					(x<COL-margine)&&(y<ROW-margine) &&
					d_image1[y][x]!=WHITE);
	// if(condition==0)return;
	double this_pixel = condition*(double)d_image1[y][x];
	__shared__ double sdata[3][TPB_X_TPB];
	sdata[0][tid] = this_pixel;
	sdata[1][tid] = this_pixel*this_pixel;
	sdata[2][tid] = condition;

	__syncthreads();

	customAdd(sdata[0],d_cuda_defcan_vars);
	customAdd(sdata[1],d_cuda_defcan_vars+1);
	customAdd(sdata[2],d_cuda_defcan_vars+2);
}
__global__ void cuda_defcan2() {
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((y >= ROW) || (x >= COL)) {
        return;
    }

	/*
		s_vars[0]:  mean
		s_vars[1]:  norm
	*/
	__shared__ double s_vars[2];
	if(threadIdx.x == 0 && threadIdx.y == 0){
    	double npo = d_cuda_defcan_vars[2];
		double mean = d_cuda_defcan_vars[0]/ (double)npo;
		double norm = d_cuda_defcan_vars[1] - (double)npo * mean * mean;
		if (norm == 0.0) norm = 1.0;
		s_vars[0] = mean;
		s_vars[1] = norm;
	}
	__syncthreads();

	int condition = ((x<COL-CANMARGIN)&&(y<ROW-CANMARGIN) &&
					d_image1[y][x]!=WHITE);
	// if(condition==0)return;
	double ratio = 1.0 / sqrt(s_vars[1]);
	d_g_can1[y][x] = ratio * ((double)d_image1[y][x] - s_vars[0]);
}

void* d_image1_ptr; void* d_image2_ptr; void* d_H_ptr;void*  d_Ht_ptr;void*  d_g_ptr;
void*  d_g_can1_ptr;void*  d_g_nor1_ptr;void*  d_g_ang1_ptr;void* d_sHoG1_ptr;
void* d_cuda_defcan_vars_ptr;
double g[G_NUM];

dim3 numBlock;
dim3 numThread;

void cuda_init_parameter(){
	numBlock.x = iDivUp(COL, TPB);
	numBlock.y = iDivUp(ROW, TPB);
	numThread.x = TPB;
	numThread.y = TPB;

	gpuErrchk( hipGetSymbolAddress(&d_image1_ptr,HIP_SYMBOL(d_image1)));
	gpuErrchk( hipGetSymbolAddress(&d_H_ptr,HIP_SYMBOL(d_H)));
	gpuErrchk( hipGetSymbolAddress(&d_Ht_ptr,HIP_SYMBOL(d_Ht)));
	gpuErrchk( hipGetSymbolAddress(&d_g_ptr,HIP_SYMBOL(d_g)));
	gpuErrchk( hipGetSymbolAddress(&d_sHoG1_ptr,HIP_SYMBOL(d_sHoG1)));
	gpuErrchk( hipGetSymbolAddress(&d_g_can1_ptr,HIP_SYMBOL(d_g_can1)));
	gpuErrchk( hipGetSymbolAddress(&d_g_nor1_ptr,HIP_SYMBOL(d_g_nor1)));
	gpuErrchk( hipGetSymbolAddress(&d_g_ang1_ptr,HIP_SYMBOL(d_g_ang1)));
	gpuErrchk( hipGetSymbolAddress(&d_cuda_defcan_vars_ptr,HIP_SYMBOL(d_cuda_defcan_vars)));

	
	gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipDeviceSynchronize() ); // Checks for execution error
	gpuErrchk( hipPeekAtLastError() ); // Checks for launch error
}

__global__ void test(){
	// int x = blockIdx.x*blockDim.x + threadIdx.x;
 //    int y = blockIdx.y*blockDim.y + threadIdx.y;
 //    if ((y >= ROW - 2 * margin) || (x >= 3 * 64 * (COL - 2 * margin))) {
 //        return;
 //    }
}

void cuda_procImg(double g_can[ROW][COL], int g_ang[ROW][COL], double g_nor[ROW][COL], char g_HoG[ROW][COL][8], char sHoG[ROW - 4][COL - 4], unsigned char image1[MAX_IMAGESIZE][MAX_IMAGESIZE]){
	hipMemset(d_cuda_defcan_vars_ptr, 0, 3 * sizeof(double));
	hipMemcpy(d_image1_ptr, image1, MAX_IMAGESIZE*MAX_IMAGESIZE*sizeof(unsigned char), hipMemcpyHostToDevice);
	numBlock.x = iDivUp(COL, TPB);
	numBlock.y = iDivUp(ROW, TPB);
	cuda_defcan1<<<numBlock, numThread>>>();
	cuda_defcan2<<<numBlock, numThread>>>();
	cuda_roberts8<<<numBlock, numThread>>>();
	hipMemcpy(g_can, d_g_can1_ptr, ROW*COL*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(g_ang, d_g_ang1_ptr, ROW*COL*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(g_nor, d_g_nor1_ptr, ROW*COL*sizeof(double), hipMemcpyDeviceToHost);
}
void cuda_calc_defcan1(double g_can1[ROW][COL], unsigned char image1[MAX_IMAGESIZE][MAX_IMAGESIZE]){
	hipMemset(d_cuda_defcan_vars_ptr, 0, 3 * sizeof(double));
	hipMemcpy(d_image1_ptr, image1, MAX_IMAGESIZE*MAX_IMAGESIZE*sizeof(unsigned char), hipMemcpyHostToDevice);
	numBlock.x = iDivUp(COL, TPB);
	numBlock.y = iDivUp(ROW, TPB);
	cuda_defcan1<<<numBlock, numThread>>>();
	cuda_defcan2<<<numBlock, numThread>>>();
	cuda_roberts8<<<numBlock, numThread>>>();
	hipMemcpy(g_can1, d_g_can1_ptr, ROW*COL*sizeof(double), hipMemcpyDeviceToHost);
}

void cuda_update_parameter(int g_ang1[ROW][COL], double g_can1[ROW][COL],double H[ROW_H][COL_H],char sHoG1[ROW - 4][COL - 4]){

	// hipMemcpy(d_g_ang1_ptr, g_ang1, ROW*COL*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sHoG1_ptr, sHoG1, (ROW - 4)*(COL-4)*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_H_ptr, H, ROW_H*COL_H*sizeof(double), hipMemcpyHostToDevice);
}

void cuda_Ht(double newVar){
	int margin = 2;
	numBlock.x = iDivUp(3 * 64 * (COL - 2 * margin), TPB);
	numBlock.y = iDivUp(ROW - 2 * margin, TPB);
	if (newVar > 1.0) {
		Ht_1<<<numBlock, numThread>>>();
	} else if (newVar < 1.0 / 32.0) {
		Ht_2<<<numBlock, numThread>>>();
	} else {
		int count = floor(log2(newVar)) + 5;
		Ht_3<<<numBlock, numThread>>>(count, newVar);
	}
	// gpuErrchk( hipDeviceSynchronize() );
 //    gpuErrchk( hipDeviceSynchronize() ); // Checks for execution error
	// gpuErrchk( hipPeekAtLastError() ); // Checks for launch error
}
double* cuda_calc_g(){
	hipMemset(d_g_ptr, 0, G_NUM * sizeof(double));
	numBlock.x = iDivUp(COL, TPB);
	numBlock.y = iDivUp(ROW, TPB);
	weightedAVG<<<numBlock, numThread>>>();
	// gpuErrchk( hipPeekAtLastError() );
 //    gpuErrchk( hipDeviceSynchronize() ); // Checks for execution error
	// gpuErrchk( hipDeviceSynchronize() );
	hipMemcpy(g, d_g_ptr, G_NUM*sizeof(double), hipMemcpyDeviceToHost);
	return g;
}
#include "hip/hip_runtime.h"
//
//  Created by Shitian Ni on 1/18/18.
//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include<iostream>
#include<string.h>
#include "parameter.h"

using namespace std;

__device__ double d_H[ROW_H][COL_H], d_Ht[ROW][COL_Ht];
__device__ unsigned char d_image1[MAX_IMAGESIZE][MAX_IMAGESIZE];
__device__ unsigned char d_image2[MAX_IMAGESIZE][MAX_IMAGESIZE];
__device__ double d_g[G_NUM], d_g_can1[ROW][COL], d_g_nor1[ROW][COL], d_gk[ROW][COL], d_gwt[ROW][COL],d_g_can2[ROW][COL];
__device__ int d_g_ang1[ROW][COL];
__device__ char d_sHoG1[ROW - 4][COL - 4];
__device__ double d_new_cor;
__device__ double d_gpt[3][3];


int iDivUp(int hostPtr, int b){ return ((hostPtr % b) != 0) ? (hostPtr / b + 1) : (hostPtr / b); };

//https://stackoverflow.com/a/14038590
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void Ht_1() {
	int margin = 2;
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((y >= ROW - 2 * margin) || (x >= 3 * 64 * (COL - 2 * margin))) {
        return;
    }

    d_Ht[y][x] =  d_H[y][x + (COL - 2 * margin) * 3 * 64 * 5];
};
__global__ void Ht_2() {
	int margin = 2;
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((y >= ROW - 2 * margin) || (x >= 3 * 64 * (COL - 2 * margin))) {
        return;
    }

    d_Ht[y][x] =  d_H[y][x];   
};
__global__ void Ht_3(int count, double newVar) {
	int margin = 2;
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((y >= ROW - 2 * margin) || (x >= 3 * 64 * (COL - 2 * margin))) {
        return;
    }
    double var_p_1 = pow(2.0,count + 1 -5);
    double var = var_p_1 / 2.0;
    d_Ht[y][x] = d_H[y][x + (COL - 2 * margin) * 3 * 64 * count] +
                                   (d_H[y][x + (COL - 2 * margin) * 3 * 64 * (count + 1)] - d_H[y][x + (COL - 2 * margin) * 3 * 64 * count])
                                 / (var_p_1 - var)
                                 * (newVar - var);                    
};


//1000 times 1200~1300ms
//http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
template<typename T>
__device__ void customAdd(T* sdata,T* g_odata){
 	int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;
	// do reduction in shared mem
	if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads();
	if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
	if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
	if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();
	if (tid < 32){ sdata[tid] += sdata[tid + 32]; }__syncthreads();
	if (tid < 16){ sdata[tid] += sdata[tid + 16]; }__syncthreads();
	if (tid < 8){ sdata[tid] += sdata[tid + 8]; }__syncthreads();
	if (tid < 4){ sdata[tid] += sdata[tid + 4]; }__syncthreads();
	if (tid < 2){ sdata[tid] += sdata[tid + 2]; }__syncthreads();
	if (tid < 1){ sdata[tid] += sdata[tid + 1]; }__syncthreads();
	// write result for this block to global mem
	if (tid == 0) {atomicAdd(g_odata        , sdata[tid]);}

}
__global__ void weightedAVG() {

	// __shared__ double sdata[TPB_X_TPB];
	__shared__ double sdata[6][TPB_X_TPB];

    int x1 = blockIdx.x*blockDim.x + threadIdx.x;
    int y1 = blockIdx.y*blockDim.y + threadIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;

    int margin = 2;

    double sHoGnumber[64] = sHoGNUMBER;
	double dx1=x1 - CX;
	double dy1=y1 - CY;
	bool condition = ((y1 >= margin) && (x1 >= margin) && (y1 < ROW-margin) && (x1 < COL-margin) && d_sHoG1[y1 - margin][x1 - margin] != -1);
	double t0 = 0;
	double tx2 = 0;
	double ty2 = 0;
	int thre = -1;
    for (int s = 0 ; condition && s < 64 ; s++) {
        if (d_sHoG1[y1 - margin][x1 - margin] == sHoGnumber[s]) {
            thre = s * 3 * (COL - 2 * margin);
            t0     = d_Ht[y1 - margin][thre + x1 - margin]                          * d_g_can1[y1][x1];
		    tx2    = d_Ht[y1 - margin][thre + x1 - margin + (COL - 2 * margin)]     * d_g_can1[y1][x1];
		    ty2    = d_Ht[y1 - margin][thre + x1 - margin + (COL - 2 * margin) * 2] * d_g_can1[y1][x1];
            break;
        }
    }
	

    sdata[0][tid]=t0; 
	sdata[1][tid]=tx2; 
	sdata[2][tid]=ty2; 
	sdata[3][tid]=t0  * dx1;
	sdata[4][tid]=t0  * dx1 * dx1; 
	sdata[5][tid]=t0  * dx1 * dx1 * dx1; 
	__syncthreads(); 

	customAdd(sdata[0],d_g); 
	customAdd(sdata[1],d_g+21);
	customAdd(sdata[2],d_g+22);
	customAdd(sdata[3],d_g+3);
	customAdd(sdata[4],d_g+4);
	customAdd(sdata[5],d_g+5);
	__syncthreads(); 

	sdata[0][tid]=t0  * dx1 * dx1 * dx1 * dx1; 
	sdata[1][tid]=t0  * dy1; 
	sdata[2][tid]=t0  * dy1 * dy1; 
	sdata[3][tid]=t0  * dy1 * dy1 * dy1;  
	sdata[4][tid]=t0  * dy1 * dy1 * dy1 * dy1; 
	sdata[5][tid]=t0  * dx1 * dy1; 
	__syncthreads(); 

	customAdd(sdata[0],d_g+6);
	customAdd(sdata[1],d_g+7);
	customAdd(sdata[2],d_g+8);
	customAdd(sdata[3],d_g+9);
	customAdd(sdata[4],d_g+10); 
	customAdd(sdata[5],d_g+11);
	__syncthreads(); 

	sdata[0][tid]=t0  * dx1 * dx1 * dy1; 
	sdata[1][tid]=t0  * dx1 * dx1 * dx1 * dy1; 
	sdata[2][tid]=t0  * dx1 * dy1 * dy1; 
	sdata[3][tid]=t0  * dx1 * dx1 * dy1 * dy1;
	sdata[4][tid]=t0  * dx1 * dy1 * dy1 * dy1;
	sdata[5][tid]=tx2 * dx1; 
	__syncthreads();  

	customAdd(sdata[0],d_g+12);
	customAdd(sdata[1],d_g+13);
	customAdd(sdata[2],d_g+14);
	customAdd(sdata[3],d_g+15);
	customAdd(sdata[4],d_g+16);
	customAdd(sdata[5],d_g+17);
	__syncthreads();  

	sdata[0][tid]=tx2 * dy1; 
	sdata[1][tid]=ty2 * dx1; 
	sdata[2][tid]=ty2 * dy1;   
	sdata[3][tid]=tx2 * dx1 * dx1; 
	sdata[4][tid]=ty2 * dx1 * dy1;  
	sdata[5][tid]=tx2 * dx1 * dy1; 
	__syncthreads();

	customAdd(sdata[0],d_g+18);
	customAdd(sdata[1],d_g+19);
	customAdd(sdata[2],d_g+20);
	customAdd(sdata[3],d_g+23);
	customAdd(sdata[4],d_g+24);
	customAdd(sdata[5],d_g+25);
	__syncthreads();

	sdata[0][tid]=ty2 * dy1 * dy1; 
	__syncthreads(); 
	customAdd(sdata[0],d_g+26); 
	__syncthreads();  
};

__global__ void cuda_roberts8() {
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((y >= ROW) || (x >= COL)) {
        return;
    }

	/* extraction of gradient information by Roberts operator */
	/* with 8-directional codes and strength */
	double delta_RD, delta_LD;
	double angle;

	/* angle & norm of gradient vector calculated
     by Roberts operator */

	if(y >= ROW-1 || x >= COL-1){
		d_g_ang1[y][x] = -1;
		d_g_nor1[y][x] = 0.0;
		return;
	}

	delta_RD = d_image1[y][x + 1] - d_image1[y + 1][x];
	delta_LD = d_image1[y][x]     - d_image1[y + 1][x + 1];
	d_g_nor1[y][x] = sqrt(delta_RD * delta_RD + delta_LD * delta_LD);

	if (d_g_nor1[y][x] == 0.0 || delta_RD * delta_RD + delta_LD * delta_LD < NoDIRECTION * NoDIRECTION) {
		d_g_ang1[y][x] = -1;
		return;
	}
	if (abs(delta_RD) == 0.0) {
		if (delta_LD > 0) d_g_ang1[y][x] = 3;
		else if (delta_LD < 0) d_g_ang1[y][x] = 7;
		else d_g_ang1[y][x] = -1;
		return;
	} 
	angle = atan2(delta_LD, delta_RD);
	if (     angle >  7.0 / 8.0 * PI) d_g_ang1[y][x] = 5;
	else if (angle >  5.0 / 8.0 * PI) d_g_ang1[y][x] = 4;
	else if (angle >  3.0 / 8.0 * PI) d_g_ang1[y][x] = 3;
	else if (angle >  1.0 / 8.0 * PI) d_g_ang1[y][x] = 2;
	else if (angle > -1.0 / 8.0 * PI) d_g_ang1[y][x] = 1;
	else if (angle > -3.0 / 8.0 * PI) d_g_ang1[y][x] = 0;
	else if (angle > -5.0 / 8.0 * PI) d_g_ang1[y][x] = 7;
	else if (angle > -7.0 / 8.0 * PI) d_g_ang1[y][x] = 6;
	else d_g_ang1[y][x] = 5;	
}

/*
	d_cuda_defcan_vars[0]:  mean
	d_cuda_defcan_vars[1]:  norm
	d_cuda_defcan_vars[2]:  npo
*/
__device__ double d_cuda_defcan_vars[3];
__global__ void cuda_defcan1() {
	int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((y >= ROW) || (x >= COL)) {
        return;
    }

	/* definite canonicalization */
	int margine = CANMARGIN / 2;
	int condition = ((x>=margine && y>=margine) && 
					(x<COL-margine)&&(y<ROW-margine) &&
					d_image1[y][x]!=WHITE);
	// if(condition==0)return;
	double this_pixel = condition*(double)d_image1[y][x];
	__shared__ double sdata[3][TPB_X_TPB];
	sdata[0][tid] = this_pixel;
	sdata[1][tid] = this_pixel*this_pixel;
	sdata[2][tid] = condition;

	__syncthreads();

	customAdd(sdata[0],d_cuda_defcan_vars);
	customAdd(sdata[1],d_cuda_defcan_vars+1);
	customAdd(sdata[2],d_cuda_defcan_vars+2);
}
__global__ void cuda_defcan2() {
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((y >= ROW) || (x >= COL)) {
        return;
    }

	/*
		s_vars[0]:  mean
		s_vars[1]:  norm
	*/
	__shared__ double s_vars[2];
	if(threadIdx.x == 0 && threadIdx.y == 0){
    	double npo = d_cuda_defcan_vars[2];
		double mean = d_cuda_defcan_vars[0]/ (double)npo;
		double norm = d_cuda_defcan_vars[1] - (double)npo * mean * mean;
		if (norm == 0.0) norm = 1.0;
		s_vars[0] = mean;
		s_vars[1] = norm;
	}
	__syncthreads();

	int condition = ((x<COL-CANMARGIN)&&(y<ROW-CANMARGIN) &&
					d_image1[y][x]!=WHITE);
	// if(condition==0)return;
	double ratio = 1.0 / sqrt(s_vars[1]);
	d_g_can1[y][x] = condition * ratio * ((double)d_image1[y][x] - s_vars[0]);
}



void* d_image1_ptr; void* d_image2_ptr; void* d_H_ptr;void*  d_Ht_ptr;void*  d_g_ptr;
void*  d_g_can1_ptr;void*  d_g_nor1_ptr;void*  d_g_ang1_ptr;void* d_sHoG1_ptr;
void* d_cuda_defcan_vars_ptr;
void* d_gk_ptr;void* d_gwt_ptr;void* d_g_can2_ptr;
void* d_new_cor_ptr;
void* d_gpt_ptr;
double g[G_NUM];
int procImg_No = 1;

dim3 numBlock;
dim3 numThread;

void cuda_init_parameter(){
	numBlock.x = iDivUp(COL, TPB);
	numBlock.y = iDivUp(ROW, TPB);
	numThread.x = TPB;
	numThread.y = TPB;

	gpuErrchk( hipGetSymbolAddress(&d_image1_ptr,HIP_SYMBOL(d_image1)));
	gpuErrchk( hipGetSymbolAddress(&d_image2_ptr,HIP_SYMBOL(d_image2)));
	gpuErrchk( hipGetSymbolAddress(&d_H_ptr,HIP_SYMBOL(d_H)));
	gpuErrchk( hipGetSymbolAddress(&d_Ht_ptr,HIP_SYMBOL(d_Ht)));
	gpuErrchk( hipGetSymbolAddress(&d_g_ptr,HIP_SYMBOL(d_g)));
	gpuErrchk( hipGetSymbolAddress(&d_sHoG1_ptr,HIP_SYMBOL(d_sHoG1)));
	gpuErrchk( hipGetSymbolAddress(&d_g_can1_ptr,HIP_SYMBOL(d_g_can1)));
	gpuErrchk( hipGetSymbolAddress(&d_g_nor1_ptr,HIP_SYMBOL(d_g_nor1)));
	gpuErrchk( hipGetSymbolAddress(&d_g_ang1_ptr,HIP_SYMBOL(d_g_ang1)));
	gpuErrchk( hipGetSymbolAddress(&d_cuda_defcan_vars_ptr,HIP_SYMBOL(d_cuda_defcan_vars)));
	gpuErrchk( hipGetSymbolAddress(&d_gk_ptr,HIP_SYMBOL(d_gk)));
	gpuErrchk( hipGetSymbolAddress(&d_gwt_ptr,HIP_SYMBOL(d_gwt)));
	gpuErrchk( hipGetSymbolAddress(&d_g_can2_ptr,HIP_SYMBOL(d_g_can2)));
	gpuErrchk( hipGetSymbolAddress(&d_new_cor_ptr, HIP_SYMBOL(d_new_cor)) );
	gpuErrchk( hipGetSymbolAddress(&d_gpt_ptr, HIP_SYMBOL(d_gpt)) );
	
	gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipDeviceSynchronize() ); // Checks for execution error
	gpuErrchk( hipPeekAtLastError() ); // Checks for launch error
}

void init_gk_and_g_can2(double gk[ROW][COL],double g_can2[ROW][COL]){
	hipMemcpy(d_gk_ptr, gk, ROW * COL * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_g_can2_ptr, g_can2, ROW * COL * sizeof(double), hipMemcpyHostToDevice);
}

__global__ void cuda_calc_gwt(double var){
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((y >= ROW) || (x >= COL)) {
        return;
    }

	d_gwt[y][x] = pow(d_gk[y][x], 1.0 / var);
}
void calc_gwt(double var,double gwt[ROW][COL]){
	numBlock.x = iDivUp(COL, TPB);
	numBlock.y = iDivUp(ROW, TPB);
	cuda_calc_gwt<<<numBlock, numThread>>>(var);
	hipMemcpy(gwt, d_gwt_ptr, ROW*COL*sizeof(double), hipMemcpyDeviceToHost);
}


__global__ void cuda_calc_new_cor1() {
	int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((y >= ROW) || (x >= COL)) {
        return;
    }

	__shared__ double sdata[TPB_X_TPB];
	sdata[tid] = d_g_can1[y][x]*d_g_can2[y][x];

	__syncthreads();

	customAdd(sdata,&d_new_cor);
}
double calc_new_cor1(){
	gpuErrchk( hipMemset(d_new_cor_ptr,0,sizeof(double)) );
	numBlock.x = iDivUp(COL, TPB);
	numBlock.y = iDivUp(ROW, TPB);
	cuda_calc_new_cor1<<<numBlock, numThread>>>();
	double new_cor;
	hipMemcpy(&new_cor, d_new_cor_ptr, sizeof(double), hipMemcpyDeviceToHost);
	return new_cor;
}

__global__ void test(){
	// int x = blockIdx.x*blockDim.x + threadIdx.x;
 //    int y = blockIdx.y*blockDim.y + threadIdx.y;
 //    if ((y >= ROW - 2 * margin) || (x >= 3 * 64 * (COL - 2 * margin))) {
 //        return;
 //    }
}

void cuda_procImg(double g_can[ROW][COL], int g_ang[ROW][COL], double g_nor[ROW][COL], char g_HoG[ROW][COL][8], char sHoG[ROW - 4][COL - 4], unsigned char image1[MAX_IMAGESIZE][MAX_IMAGESIZE]){
	hipMemset(d_cuda_defcan_vars_ptr, 0, 3 * sizeof(double));
	if(procImg_No == 1)
		hipMemcpy(d_image1_ptr, image1, MAX_IMAGESIZE*MAX_IMAGESIZE*sizeof(unsigned char), hipMemcpyHostToDevice);
	numBlock.x = iDivUp(COL, TPB);
	numBlock.y = iDivUp(ROW, TPB);
	cuda_defcan1<<<numBlock, numThread>>>();
	cuda_defcan2<<<numBlock, numThread>>>();
	cuda_roberts8<<<numBlock, numThread>>>();
	hipMemcpy(g_can, d_g_can1_ptr, ROW*COL*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(g_ang, d_g_ang1_ptr, ROW*COL*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(g_nor, d_g_nor1_ptr, ROW*COL*sizeof(double), hipMemcpyDeviceToHost);

	procImg_No = max(procImg_No+1,2);
}
void cuda_calc_defcan1(double g_can1[ROW][COL], unsigned char image1[MAX_IMAGESIZE][MAX_IMAGESIZE]){
	// hipMemset(d_cuda_defcan_vars_ptr, 0, 3 * sizeof(double));
	// // hipMemcpy(d_image1_ptr, image1, MAX_IMAGESIZE*MAX_IMAGESIZE*sizeof(unsigned char), hipMemcpyHostToDevice);
	// numBlock.x = iDivUp(COL, TPB);
	// numBlock.y = iDivUp(ROW, TPB);
	// cuda_defcan1<<<numBlock, numThread>>>();
	// cuda_defcan2<<<numBlock, numThread>>>();
	// cuda_roberts8<<<numBlock, numThread>>>();
	// hipMemcpy(g_can1, d_g_can1_ptr, ROW*COL*sizeof(double), hipMemcpyDeviceToHost);
}

int needH = 1;
void cuda_update_parameter(int g_ang1[ROW][COL], double g_can1[ROW][COL],double H[ROW_H][COL_H],char sHoG1[ROW - 4][COL - 4]){

	// hipMemcpy(d_g_ang1_ptr, g_ang1, ROW*COL*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sHoG1_ptr, sHoG1, (ROW - 4)*(COL-4)*sizeof(char), hipMemcpyHostToDevice);
	if(needH)
		hipMemcpy(d_H_ptr, H, ROW_H*COL_H*sizeof(double), hipMemcpyHostToDevice);
	needH = max(0,needH-1);
}

void cuda_Ht(double newVar){
	int margin = 2;
	numBlock.x = iDivUp(3 * 64 * (COL - 2 * margin), TPB);
	numBlock.y = iDivUp(ROW - 2 * margin, TPB);
	if (newVar > 1.0) {
		Ht_1<<<numBlock, numThread>>>();
	} else if (newVar < 1.0 / 32.0) {
		Ht_2<<<numBlock, numThread>>>();
	} else {
		int count = floor(log2(newVar)) + 5;
		Ht_3<<<numBlock, numThread>>>(count, newVar);
	}
	// gpuErrchk( hipDeviceSynchronize() );
 //    gpuErrchk( hipDeviceSynchronize() ); // Checks for execution error
	// gpuErrchk( hipPeekAtLastError() ); // Checks for launch error
}
double* cuda_calc_g(){
	hipMemset(d_g_ptr, 0, G_NUM * sizeof(double));
	numBlock.x = iDivUp(COL, TPB);
	numBlock.y = iDivUp(ROW, TPB);
	weightedAVG<<<numBlock, numThread>>>();
	// gpuErrchk( hipPeekAtLastError() );
 //    gpuErrchk( hipDeviceSynchronize() ); // Checks for execution error
	// gpuErrchk( hipDeviceSynchronize() );
	hipMemcpy(g, d_g_ptr, G_NUM*sizeof(double), hipMemcpyDeviceToHost);
	return g;
}

__device__ void cuda_multiplyVect3x3(double inMat[3][3], double inVect[3], double outVect[3]) {
	int i, j;
	double sum;
	for(i = 0 ; i < 3 ; ++i) {
		sum = 0.0;
		for(j = 0 ; j < 3 ; ++j) {
			sum += inMat[i][j] * inVect[j];
		}
		outVect[i] = sum;
	}
}

__global__ void cuda_calc_bilinear_normal_inverse_projection(int x_size1, int y_size1, int x_size2, int y_size2){
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((y >= y_size1) || (x >= x_size1)) {
        return;
    }
	int cx, cy, cx2, cy2;
    if (y_size1 == ROW) {
		cx  = CX,  cy  = CY;
		cx2 = CX2, cy2 = CY2;
	} else {
		cx  = CX2, cy  = CY2;
		cx2 = CX,  cy2 = CY;
	}

    double inVect[3], outVect[3];
	double x_new, y_new, x_frac, y_frac;
	double gray_new;
	int m, n;
	
	inVect[2] = 1.0;
	inVect[1] = y - cy;
	inVect[0] = x - cx;

	int i, j;
	double sum;
	for(i = 0 ; i < 3 ; ++i) {
		sum = 0.0;
		for(j = 0 ; j < 3 ; ++j) {
			sum += d_gpt[i][j] * inVect[j];
		}
		outVect[i] = sum;
	}

	x_new = outVect[0] / outVect[2] + cx2;
	y_new = outVect[1] / outVect[2] + cy2;
	m = (int)floor(x_new);
	n = (int)floor(y_new);
	x_frac = x_new - m;
	y_frac = y_new - n;

	if (m >= 0 && m+1 < x_size2 && n >= 0 && n+1 < y_size2) {
		gray_new = (1.0 - y_frac) * ((1.0 - x_frac) * d_image1[n][m] + x_frac * d_image1[n][m+1])
		 + y_frac * ((1.0 - x_frac) * d_image1[n+1][m] + x_frac * d_image1[n+1][m+1]);
		d_image2[y][x] = (unsigned char)gray_new;
	} else {
	#ifdef BACKGBLACK
		d_image2[y][x] = BLACK;
	#else
		d_image2[y][x] = WHITE;
	#endif
	}
}

__global__ void image2_to_image1(){
	int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if ((y >= MAX_IMAGESIZE) || (x >= MAX_IMAGESIZE)) {
        return;
    }
    d_image1[y][x] = d_image2[y][x];
}

void cuda_bilinear_normal_inverse_projection(double gpt[3][3], int x_size1, int y_size1, int x_size2, int y_size2,
		unsigned char image1[MAX_IMAGESIZE][MAX_IMAGESIZE], unsigned char image2[MAX_IMAGESIZE][MAX_IMAGESIZE]) {
	/* inverse projection transformation of the image by bilinear interpolation */
	numBlock.x = iDivUp(x_size1, TPB);
	numBlock.y = iDivUp(y_size1, TPB);
	hipMemcpy(d_image1_ptr,image1,MAX_IMAGESIZE*MAX_IMAGESIZE*sizeof(unsigned char),hipMemcpyHostToDevice);
	hipMemcpy(d_gpt_ptr,gpt,3*3*sizeof(double),hipMemcpyHostToDevice);

	cuda_calc_bilinear_normal_inverse_projection<<<numBlock, numThread>>>(x_size1, y_size1, x_size2, y_size2);
	hipMemcpy(image2, d_image2_ptr, MAX_IMAGESIZE*MAX_IMAGESIZE*sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipMemcpy(d_image1_ptr, d_image2_ptr, MAX_IMAGESIZE*MAX_IMAGESIZE*sizeof(unsigned char), hipMemcpyDeviceToHost);
}